#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdlib.h>
#include <string.h>

#include <cutil.h>
#include "util.h"
#include "ref_2dhisto.h"

//__global__ void histoKernel(uint32_t*, size_t, size_t, uint32_t*);
//__global__ void opt_32to8Kernel(uint32_t*, uint8_t*, size_t);

__global__ void histoKernel2(uint32_t *input, size_t height, uint32_t* bins);
__global__ void saturate(unsigned int *bins, unsigned int num_bins);

void opt_2dhisto(uint32_t* input, size_t height, size_t width, uint8_t* bins, uint32_t* g_bins)
{
    /* This function should only contain a call to the GPU
       histogramming kernel. Any memory allocations and
       transfers must be done outside this function */


    //Creating 32_bit histogram in parallel
    //blockDim is 32*32 = 1024 threads per block
    //gridDim is (size of the input data) / (blockDim)

    histoKernel<<<INPUT_HEIGHT * ((INPUT_WIDTH + 128) & 0xFFFFFF80) / 1024 , 1024>>>(input, height, width, g_bins);

    //Converting 32_bit histogram to 8 bit
    opt_32to8Kernel<<<HISTO_HEIGHT * HISTO_WIDTH / 1024, 1024>>>(g_bins, bins, 1024);
    hipDeviceSynchronize();
}

/* Include below the implementation of any other functions you need */
/*
__global__ void histoKernel(uint32_t *input, size_t height, size_t width, uint32_t* bins){
      int globalTid = blockDim.x * blockIdx.x + threadIdx.x;
     //__shared__ uint32_t* s_input = input;
     if (globalTid < 1024)
        bins[globalTid] = 0;
     __syncthreads();
     int stride = blockDim.x * gridDim.x;
     while (globalTid < 4096 * height)
     {
        if (globalTid %  ((INPUT_WIDTH + 128) & 0xFFFFFF80) < width )
           atomicAdd( &(bins[input[globalTid]]), 1 );
        globalTid += stride;
     }
}

__global__ void opt_32to8Kernel(uint32_t *input, uint8_t* output, size_t length){
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	output[idx] = (uint8_t)((input[idx] < UINT8_MAX) * input[idx]) + (input[idx] >= UINT8_MAX) * UINT8_MAX;
	__syncthreads();
}
*/

__global__ void histoKernel2(uint32_t *input, size_t height, uint32_t* bins){
  int globalTid = blockDim.x * blockIdx.x + threadIdx.x;
  unsigned int num_elements = height * width;
  __shared__ unsigned int s_bins[num_elements];
  int stride = blockDim.x * gridDim.x;
  if (threadIdx.x < num_bins) {
		s_bins[threadIdx.x] = 0;
	}
  while (globalTid < num_elements){
     int value = input[globalTid];
     atomicAdd( &(s_bins[value]), 1);
     globalTid += stride;
  }
  __syncthreads();

  if (threadIdx.x < num_bins) {
  		atomicAdd(&(bins[threadIdx.x]), s_bins[threadIdx.x]);
  	}
}

__global__ void saturate(unsigned int *bins, unsigned int num_bins) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;

	if (i < num_bins) {
		if (bins[i] > 255) {
			bins[i] = 255;
    }
	}
}

void* AllocateOnDevice(size_t size){
	void* ret;
	hipMalloc(&ret, size);
	return ret;
}

void CopyToDevice(void* d_device, void* d_host, size_t size){
	hipMemcpy(d_device, d_host, size,hipMemcpyHostToDevice);
}

void CopyFromDevice(void* d_host, void* d_device, size_t size){
	hipMemcpy(d_host, d_device, size,hipMemcpyDeviceToHost);
}

void FreeCuda(void* d_space){
	hipFree(d_space);
}
